#include "hip/hip_runtime.h"
/*
 * kernel.cu
 *
 *  Created on: 26/01/2019
 *      Author: minterciso
 */
#include "kernels.h"
#include "prng.h"
#include "consts.h"
#include "utils.h"

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hiprand/hiprand_kernel.h>
#include <time.h>
#include <sys/time.h>
#include <gsl/gsl_rng.h>


/*********
 * ROBBY *
 ********/
__global__ void execute_population(hiprandState *states, int amount_states, robby *d_robby, int amount_robby/*, world *d_world*/, int amount_world){
  const int tid = threadIdx.x + blockIdx.x*blockDim.x;
  if(tid < amount_states && tid < amount_robby){
    for(int world_id = 0; world_id < amount_world; world_id++){
      int score = 0;
      world world;;
      create_world(&states[tid], &world);
      for(int i=0;i<200;i++)
        score += execute_strategy(&states[tid], &d_robby[tid], &world);
      d_robby[tid].fitness += (float)score;
    }
    d_robby[tid].fitness /= amount_world;
  }
}

__device__ int to_decimal(int *arr, int base, int len){
  int power = 1;
  int num = 0;
  int i;
  for(i=len-1;i>=0;i--){
    if(arr[i] >= base)
      return -1;
    num += (arr[i] * power);
    power *= base;
  }
  return num;
}

__global__ void create_population(hiprandState *states, int amount_states, robby *d_robby, int amount_robby){
  const int tid = threadIdx.x + blockDim.x*blockIdx.x;
  if(tid < amount_states && tid < amount_robby){
    hiprandState local_state = states[tid];
    d_robby[tid].fitness = -99.0;
    d_robby[tid].weight = -99.0;
    for(int i=0;i<S_SIZE;i++)
      //d_robby[tid].strategy[i] = get_uniform(&local_state, 0, S_MAX_OPTIONS);
      d_robby[tid].strategy[i] = (int)(hiprand_uniform(&local_state) * S_MAX_OPTIONS);
    states[tid] = local_state;
  }
}

__device__ int execute_strategy(hiprandState *state, robby *d_robby, world *w){
  int n[5]; // neighbours
  int r_row, r_col; // robby position
  int strategy_id;
  int strategy_step;
  int movement;
  int score = 0;

  // Get robby position
  r_row = w->r_row;
  r_col = w->r_col;

  // Get neighbours
  if(r_row == 0)
    n[0] = T_WALL;
  else
    n[0] = w->tiles[r_row-1][r_col];
  if(r_row == W_ROWS - 1)
    n[1] = T_WALL;
  else
    n[1] = w->tiles[r_row+1][r_col];
  if(r_col == 0)
    n[2] = T_WALL;
  else
    n[2] = w->tiles[r_row][r_col-1];
  if(r_col == W_COLS - 1)
    n[3] = T_WALL;
  else
    n[3] = w->tiles[r_row][r_col+1];
  n[4] = w->tiles[r_row][r_col];

  // Now get the id of the strategy
  strategy_id = to_decimal(n, 3, 5);
  strategy_step = d_robby->strategy[strategy_id];
  if(strategy_step == S_RANDOM)
    movement = hiprand_uniform(state) * 4;
  else
    movement = strategy_step;
  switch(movement){
    case S_MOVE_NORTH:
      if(r_row == 0)
        score = -5;
      else
        w->r_row--;
      break;
    case S_MOVE_SOUTH:
      if(r_row == W_ROWS-1)
        score = -5;
      else
        w->r_row++;
      break;
    case S_MOVE_WEST:
      if(r_col == 0)
        score = -5;
      else
        w->r_col--;
      break;
    case S_MOVE_EAST:
      if(r_col == W_COLS - 1)
        score = -5;
      else
        w->r_col++;
      break;
    case S_STAY_PUT:
      score = 0;
      break;
    case S_PICK_UP:
      if(w->tiles[r_row][r_col] == T_CAN){
          score = 10;
          w->tiles[r_row][r_col] = T_EMPTY;
          w->qtd_cans--;
      }
      else
        score = -1;
      break;
  }

  return score;
}

/********
 * PRNG *
 *******/
__global__ void setup_prng(hiprandState *state, unsigned long long seed, unsigned int amount){
  const int tid = threadIdx.x + blockDim.x*blockIdx.x;
  if(tid < amount)
    hiprand_init(seed, 0, tid, &state[tid]);
}

__global__ void test_prng(hiprandState *state, unsigned int state_amnt, float *data, int data_amount){
  const int tid = threadIdx.x + blockDim.x * blockIdx.x;
  if(tid < state_amnt && tid < data_amount){
    float sum = 0.0;
    hiprandState local_state = state[tid];
    for(int i=0;i<100;i++)
      sum += hiprand_uniform(&local_state);
    data[tid] = sum/100;
    state[tid] = local_state;
  }
}

__device__ int get_uniform(hiprandState *state, int min, int max){
  return (int)( min + hiprand_uniform(state) * max);
}

__device__ float get_uniform(hiprandState *state){
  return hiprand_uniform(state);
}

__global__ void test_prng_uniform(hiprandState *states, unsigned int state_amnt, int *data, int data_amount, int min, int max){
  const int tid = threadIdx.x + blockDim.x * blockIdx.x;
  if(tid < state_amnt && tid < data_amount){
    hiprandState local_state = states[tid];
    data[tid] = get_uniform(&local_state, min, max);
    states[tid] = local_state;
  }
}

__global__ void test_prng_uniform(hiprandState *states, unsigned int state_amnt, float *data, int data_amount){
  const int tid = threadIdx.x + blockDim.x * blockIdx.x;
  if(tid < state_amnt && tid < data_amount){
    hiprandState local_state = states[tid];
    data[tid] = hiprand_uniform(&local_state);
    states[tid] = local_state;
  }
}

/*********
 * WORLD *
 ********/
__global__ void create_worlds(hiprandState *states, int amount_states, world* d_worlds, int amount_worlds){
  const int state_id = threadIdx.x + blockIdx.x*blockDim.x;
  if(state_id < amount_states && state_id < amount_worlds){
    hiprandState local_state = states[state_id];
    create_world(&local_state, &d_worlds[state_id]);
    states[state_id] = local_state;
  }
}

__device__ int create_world(hiprandState *state, world *d_world){
  d_world->qtd_cans = 0;
  d_world->r_row = 0;
  d_world->r_col = 0;
  for(int i=0;i<W_ROWS;i++){
    for(int j=0;j<W_COLS;j++){
      if(hiprand_uniform(state) < P_CAN){
        d_world->tiles[i][j] = T_CAN;
        d_world->qtd_cans++;
      }
      else{
        d_world->tiles[i][j] = T_EMPTY;
      }
    }
  }
  return d_world->qtd_cans;
}
