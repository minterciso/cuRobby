#include "hip/hip_runtime.h"
/*
 * =====================================================================================
 *
 *       Filename:  world.cu
 *
 *    Description:  This defines the world of Robby. It needs to:
 *    - Describe the world for Robby
 *    - One Kernel to create a lot of worlds on the Device
 *    - One device function to create one world on the Device
 *
 *        Version:  1.0
 *        Created:  17/01/2019 17:30:49
 *       Revision:  none
 *       Compiler:  nvcc
 *
 *         Author:  Mateus Interciso (mi), minterciso@gmail.com
 *        Company:  Geekvault
 *
 * =====================================================================================
 */
#include "world.h"
#include "prng.h"

#include <stdio.h>

__global__ void create_worlds(hiprandState *states, int amount_states, world* d_worlds, int amount_worlds){
  const int state_id = threadIdx.x + blockIdx.x*blockDim.x;
  if(state_id < amount_states){
    hiprandState local_state = states[state_id];
    for(int i=0;i<amount_worlds;i++){
      for(int j=0;j<W_ROWS;j++){
        for(int k=0;k<W_COLS;k++){
          if(hiprand_uniform(&local_state) < P_CAN){
            d_worlds[i].tiles[j][k] = T_CAN;
            d_worlds[i].qtd_cans++;
          }
        }
      }
    }
    states[state_id] = local_state;
  }
  /*
  const int world_id = threadIdx.y + blockIdx.y*blockDim.y;
  if(state_id < amount_states && world_id < amount_worlds){
    hiprandState local_state = states[state_id];
    float rnd = 0.0f;
    int idx = 0;
    for(int j=0;j<W_ROWS;j++){
      for(int k=0;k<W_COLS;k++){
        rnd = hiprand_uniform(&local_state);
        if(world_id == 0)
          printf("%d - rnd=%.2f\n", idx++, rnd);
        if(rnd < P_CAN){
          //if(get_uniform(&local_state) <= P_CAN){
          d_worlds[world_id].tiles[j][k] = T_CAN;
          d_worlds[world_id].qtd_cans++;
        }
        else
          d_worlds[world_id].tiles[j][k] = T_EMPTY;
      }
    }    
    states[state_id] = local_state;
  }
  */
}

__device__ int create_world(hiprandState *state, world *d_world){
  return 0;
}

__device__ int reset_world(hiprandState *state, world *d_world){
  return 0;
}
