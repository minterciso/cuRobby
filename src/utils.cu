#include "utils.h"

void start_device(void){
    CUDA_CALL(hipSetDevice(0));
}

void reset_device(void){
    CUDA_CALL(hipDeviceReset());
}
