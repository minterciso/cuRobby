#include "utils.h"

int start_device(void){
    CUDA_CALL(hipSetDevice(0));
}

int reset_device(void){
    CUDA_CALL(hipDeviceReset());
}
